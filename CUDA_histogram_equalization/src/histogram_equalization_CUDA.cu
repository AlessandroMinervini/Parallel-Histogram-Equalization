#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : histogram_equalization_CUDA.cu
 Author      : francesco
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <stdlib.h>

#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

using namespace std;
using namespace cv;

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void make_histogram(unsigned char *image, int width, int height, int *histogram){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	long index;

	for(int i = idx; i < width * height; i += blockDim.x * gridDim.x){

		index = i * 3;

		int R = image[index];
		int G = image[index + 1];
		int B = image[index + 2];

		int Y = R * .299000 + G * .587000 + B * .114000;
		int U = R * -.168736 + G * -0.331264 + B * .500000 + 128;
		int V = R * .500000 + G * -.418688 + B * -.081312 + 128;

		atomicAdd(&(histogram[Y]),1);

		image[index] = Y;
		image[index + 1] = U;
		image[index + 2] = V;
	}

	__syncthreads();
}

__global__ void equalize(int *equalized, int *cumulative_dist, int *histogram, int width, int height){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for(int k = idx; k < 256; k += blockDim.x * gridDim.x){
		equalized[k] = (int)(((float)cumulative_dist[k] - histogram[0])/((float)width * height - 1) * 255);
	}
}

__global__ void YUV2RGB(unsigned char *image, int *cumulative_dist,int *histogram, int *equalized, int width, int height){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	long index;

	for(int i = idx; i < width * height; i += blockDim.x * gridDim.x){

		index = i * 3;

		int Y = equalized[image[index]];
		int U = image[index + 1];
		int V = image[index + 2];

		unsigned char R = (unsigned char)max(0, min(255,(int)(Y + 1.4075 * (V - 128))));
		unsigned char G = (unsigned char)max(0, min(255,(int)(Y - 1.3455 * (U - 128) - (.7169 * (V - 128)))));
		unsigned char B = (unsigned char)max(0, min(255,(int)(Y + 1.7790 * (U - 128))));

		image[index] = R;
		image[index + 1] = G;
		image[index + 2] = B;

	}

}

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err){
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

int main(){

	Mat image = imread("src/images/desk.jpg");		//load the image

	if(!image.data){
		cout << "no image found";
		return -1;
	}

	int width = image.cols;
	int height = image.rows;

	int host_histogram[256];						//cpu histogram
	int host_equalized[256];						//cpu equalized histogram
	int host_cumulative_dist[256];

	unsigned char *host_image = image.ptr();		//Mat image to array image

	for(int i = 0; i < 256; i++){
		host_histogram[i] = 0;
	}

	unsigned char *device_image;	//gpu image

	int *device_histogram;			//gpu histogram
	int *device_equalized;			//gpu equalized histogram
	int *device_cumulative_dist;	//gpu cumulative dist.

	CUDA_CHECK_RETURN(hipMalloc(&device_image, sizeof(char) * (width * height * 3)));										//gpu space allocation
	CUDA_CHECK_RETURN(hipMalloc(&device_histogram, sizeof(int) * 256));													//
	CUDA_CHECK_RETURN(hipMalloc(&device_equalized, sizeof(int) * 256));													//
	CUDA_CHECK_RETURN(hipMalloc(&device_cumulative_dist, sizeof(int) * 256));												//

	CUDA_CHECK_RETURN(hipMemcpy(device_image, host_image, sizeof(char) * (width * height * 3), hipMemcpyHostToDevice));	//copy to gpu
	CUDA_CHECK_RETURN(hipMemcpy(device_histogram, host_histogram, sizeof(int) * 256, hipMemcpyHostToDevice));				//

	int block_size = 256;
	int grid_size = (width * height + (block_size - 1))/block_size;

	make_histogram<<<grid_size, block_size>>> (device_image, width, height, device_histogram);		//call first kernel

	CUDA_CHECK_RETURN(hipMemcpy(host_histogram, device_histogram, sizeof(int) * 256, hipMemcpyDeviceToHost));

	host_cumulative_dist[0] = host_histogram[0];										//compute cumulative dist. in cpu
																						//
	for(int i = 1; i < 256; i++){														//
		host_cumulative_dist[i] = host_histogram[i] + host_cumulative_dist[i-1];		//
	}																					//

	CUDA_CHECK_RETURN(hipMemcpy(device_cumulative_dist, host_cumulative_dist, sizeof(int) * 256, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(device_equalized, host_equalized, sizeof(int) * 256, hipMemcpyHostToDevice));

	equalize<<<grid_size, block_size>>>(device_equalized, device_cumulative_dist, device_histogram, width, height);					//call second kernel

	YUV2RGB<<<grid_size, block_size>>>(device_image, device_cumulative_dist, device_histogram, device_equalized, width, height);	//call third kernel

	CUDA_CHECK_RETURN(hipMemcpy(host_image, device_image, sizeof(char) * (width * height * 3), hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree(device_image));						//free gpu
	CUDA_CHECK_RETURN(hipFree(device_histogram));					//
	CUDA_CHECK_RETURN(hipFree(device_equalized));					//
	CUDA_CHECK_RETURN(hipFree(device_cumulative_dist));			//

	cout << "correctly freed memory \n";

	Mat final_image = Mat(Size(width,height), CV_8UC3, host_image);
	imwrite("src/saved/desk.jpg", final_image);						//save equalized RGB image
	cout << "correctly saved image";

	return 0;

}
